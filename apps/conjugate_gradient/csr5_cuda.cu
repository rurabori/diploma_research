#include "cuda_algo_facade.h"
#include <anonymouslib_cuda.cuh>
#include <cuda_interop/annotations.h>
#include <cuda_interop/memory.h>
#include <fmt/format.h>

__host__ void bench_csr5_cuda(int rows, int cols, int non_zero, int* row_start_offsets, int* col_indices,
                              double* values, double* rhs, double* output) {
    // set device
    int device_id = 0;
    hipSetDevice(device_id);
    hipDeviceProp_t deviceProp{};
    hipGetDeviceProperties(&deviceProp, device_id);

    fmt::print("Device [{}] {}, @{}MHz\n", device_id, deviceProp.name, deviceProp.clockRate * 1e-3f);

    auto d_row_start_offsets = cui::device_create(row_start_offsets, rows + 1);
    auto d_col_indices = cui::device_create(col_indices, non_zero);
    auto d_values = cui::device_create(values, non_zero);

    anonymouslibHandle<int, unsigned int, double> A(rows, cols);
    A.inputCSR(non_zero, d_row_start_offsets.get(), d_col_indices.get(), d_values.get());

    auto d_x = cui::device_create(rhs, cols);
    A.setX(d_x.get()); // you only need to do it once!

    A.setSigma(ANONYMOUSLIB_AUTO_TUNED_SIGMA);

    anonymouslib_timer asCSR5_timer;
    asCSR5_timer.start();
    A.asCSR5();
    fmt::print("CSR->CSR5 time = {}ms.\n", asCSR5_timer.stop());

    auto d_y = cui::alloc<double>(rows);
    anonymouslib_timer spmv_timer;
    spmv_timer.start();
    A.spmv(1.0, d_y.get());
    fmt::print("SpMV time = {}ms.\n", spmv_timer.stop());

    cui::memcpy(output, d_y.get(), rows, hipMemcpyDeviceToHost);
}